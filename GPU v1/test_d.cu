#include "test_d.cuh"

test_d::test_d(test t) : anwser(t.anwser)
{
	hipMalloc(&features, t.features.size() * sizeof(double));
	hipMemcpy(features, &t.features[0], t.features.size() * sizeof(double), hipMemcpyHostToDevice);
}

//test_d::~test_d()
//{
//	hipFree(features);
//}
