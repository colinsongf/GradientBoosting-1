#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <ctime>
#include "tree.cuh"
#include "thrust\device_vector.h"

int main()
{
	//hipDeviceSetLimit(hipLimitMallocHeapSize, 10000*1024*sizeof(node));
	clock_t time = clock();
	data_set train_set("Prototask.train4", 21, false);
	tree t(train_set, 1000000);
	/*data_set test_set("Prototask.test", false);
	double error = t.calculate_error(test_set);
	std::cout << "test error: " << error << std::endl;
	t.print();*/
	time = clock() - time;
	printf("time: %f\n\n", (double)time / CLOCKS_PER_SEC);
	hipDeviceReset();
	return 0;
}