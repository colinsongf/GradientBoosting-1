#include "test_d.cuh"

test_d::test_d(test t)
{
	hipMalloc(&anwser, sizeof(double));
	hipMemcpy(anwser, &t.anwser, sizeof(double), hipMemcpyHostToDevice);
	hipMalloc(&features, t.features.size() * sizeof(double));
	hipMemcpy(features, &t.features[0], t.features.size() * sizeof(double), hipMemcpyHostToDevice);
}

test_d::test_d() {}


test_d::~test_d()
{
	/*hipFree(anwser);
	hipFree(features);*/
}
