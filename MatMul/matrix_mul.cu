#include "hip/hip_runtime.h"

#include <cstdlib>
#include <ctime>
#include <cstdio>
#include <iostream>
#include <hipblas.h>

#define BLOCK_SIZE 32

void matMulCpu(float* a, float* b, float* c, int s1, int s2, int s3)
{
	for (int i = 0; i < s1; i++)
	{
		for (int j = 0; j < s3; j++)
		{
			float ans = 0;
			for (int k = 0; k < s2; k++)
			{
				ans += a[i * s2 + k] * b[k * s3 + j];
			}
			c[i * s3 + j] = ans;
		}
	}
}

__global__ void matMulGpu(float* a, float* b, float* c, int s1, int s2, int s3)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < s1 && j < s3)
	{
		float ans = 0;
		for (int k = 0; k < s2; k++)
		{
			ans += a[i * s2 + k] * b[k * s3 + j];
		}
		c[i * s3 + j] = ans;
	}
}

void fillMat(float* a, int s)
{
	for (int i = 0; i < s; i++)
	{
		//a[i] = rand() % 100;
		a[i] = (0.8 * ((float)rand() / (float)RAND_MAX) + 0.1);
	}
}

float calcSumCpu(float* a, int s)
{
	float ans = 0;
	for (int i = 0; i < s; i++)
	{
		ans += a[i];
	}
	return ans;
}

int main()
{
	//freopen("out.txt", "w", stdout);
	int iterations = 1;
	int size = 5000;
	srand(time(NULL));
	float* a = (float*)malloc(size * size * sizeof(float));
	float* b = (float*)malloc(size * size * sizeof(float));
	float* c = (float*)malloc(size * size * sizeof(float));

	float* a_device;
	float* b_device;
	float* c_device;
	float* c_device_cublas;
	hipMalloc(&a_device, size * size * sizeof(float));
	hipMalloc(&b_device, size * size * sizeof(float));
	hipMalloc(&c_device, size * size * sizeof(float));
	hipMalloc(&c_device_cublas, size * size * sizeof(float));

	int grid_size = 1 + size / BLOCK_SIZE;

	dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
//	dim3 grid(2 + size / (1 + BLOCK_SIZE), 2 + size / (1 + BLOCK_SIZE), 1);
	dim3 grid(grid_size, grid_size, 1);

	float sum_h;
	float sum_d;

	clock_t time_h = 0;
	float time_d_event = 0;
	float time_d_cublas = 0;

	hipblasHandle_t handle;
	hipblasStatus_t status = hipblasCreate(&handle); 
	float alpha = 1.0f;
	float beta = 0.0f;

	// warmup
	fillMat(a, size * size);
	fillMat(b, size * size);
	hipMemcpy(a_device, a, size * size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b_device, b, size * size * sizeof(float), hipMemcpyHostToDevice);
	matMulGpu<<<grid, block>>>(a_device, b_device, c_device, size, size, size);
	hipDeviceSynchronize();
	 hipError_t error = hipGetLastError();
	status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, size, size, size, &alpha, b_device, size, a_device, size,
			&beta, c_device_cublas, size);
	hipDeviceSynchronize();
	// done

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for (int i = 0; i < iterations; i++)
	{
		fillMat(a, size * size);
		fillMat(b, size * size);
		hipMemcpy(a_device, a, size * size * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(b_device, b, size * size * sizeof(float), hipMemcpyHostToDevice);

		clock_t t1 = clock();
		float time;
		matMulCpu(a, b, c, size, size, size);
		t1 = clock() - t1;
		time_h += t1;

		hipEventRecord(start, 0);
		matMulGpu<<<grid, block>>>(a_device, b_device, c_device, size, size, size);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		
		hipEventElapsedTime(&time, start, stop);
		time_d_event += (time / 1000.0);

		hipEventRecord(start, 0);
		status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, size, size, size, &alpha, b_device, size, a_device, size,
			&beta, c_device_cublas, size);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		time_d_cublas += (time / 1000.0);

		sum_h = calcSumCpu(c, size * size);
		hipMemcpy(c, c_device, size * size * sizeof(float), hipMemcpyDeviceToHost);
		sum_d = calcSumCpu(c, size * size);
		if (sum_h == sum_d)
		{
			printf("OK ");
		}
		printf("host: %f device: %f ", sum_h, sum_d);
		hipMemcpy(c, c_device_cublas, size * size * sizeof(float), hipMemcpyDeviceToHost);
		sum_d = calcSumCpu(c, size * size);
		if (sum_h == sum_d)
		{
			printf("OK ");
		}
		printf("cublas: %f\n", sum_d);
	}

	float time_h_secs = (float)time_h / CLOCKS_PER_SEC;
	float profit_event = time_h_secs / time_d_event;
	float profit_cublas = time_h_secs / time_d_cublas;
	printf("profit event: %f profit cublas %f time_h: %f time_d_event: %f time_d_cublas: %f\n\n", 
		profit_event, profit_cublas, time_h_secs, time_d_event, time_d_cublas);

	//fclose(stdout);
	status = hipblasDestroy(handle);
	free(a);
	free(b);
	free(c);
	hipFree(a_device);
	hipFree(b_device);
	hipFree(c_device);

    return 0;
}