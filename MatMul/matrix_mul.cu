#include "hip/hip_runtime.h"

#include <cstdlib>
#include <ctime>
#include <cstdio>
#include <iostream>
#include <hipblas.h>

void matrix_mul_cpu(float* a, float* b, float* c, int a_height, int a_width, int b_width) // a * b = c; 
{
	for (int i = 0; i < a_height; i++)
	{
		for (int j = 0; j < b_width; j++)
		{
			float ans = 0;
			for (int k = 0; k < a_width; k++)
			{
				ans += a[i * a_width + k] * b[k * b_width + j];
			}
			c[i * b_width + j] = ans;
		}
	}
}


void fill_matrix(float* a, int size)
{
	for (int i = 0; i < size; i++)
	{
		a[i] = rand() % 100;
	}
}

void print_matrix(float* a, int height, int width)
{
	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			printf("%f ", a[i * width + j]);
		}
		printf("\n");
	}
	printf("\n");
}

float calculate_sum_cpu(float* a, int size)
{
	float ans = 0;
	for (int i = 0; i < size; i++)
	{
		ans += a[i];
	}
	return ans;
}

int main()
{
	//freopen("out.txt", "w", stdout);
	int iterations = 1;
	//int size = 1000;
	int a_height = 500;
	int a_width = 700;
	int b_height = 700;
	int b_width = 800;
	srand(time(NULL));
	float* a = (float*)malloc(a_height * a_width * sizeof(float));
	float* b = (float*)malloc(b_height * b_width * sizeof(float));
	float* c = (float*)malloc(a_height * b_width * sizeof(float));

	float* a_device;
	float* b_device;
	float* c_device;
	hipMalloc(&a_device, a_height * a_width * sizeof(float));
	hipMalloc(&b_device, b_height * b_width * sizeof(float));
	hipMalloc(&c_device, a_height * b_width * sizeof(float));
	
	float sum_h;
	float sum_d;

	clock_t time_h = 0;
	float time_d = 0;
	
	hipblasHandle_t handle;
	hipblasStatus_t status = hipblasCreate(&handle);
	float alpha = 1.0f;
	float beta = 0.0f;

	// warmup
	fill_matrix(a, a_height * a_width);
	fill_matrix(b, b_height * b_width);
	hipMemcpy(a_device, a, a_height * a_width * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b_device, b, b_height * b_width * sizeof(float), hipMemcpyHostToDevice);
		status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, b_width, a_height, a_width, &alpha, b_device, b_width, a_device,
			a_width, &beta, c_device, b_width);
	hipDeviceSynchronize();
	// done

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for (int i = 0; i < iterations; i++)
	{
		fill_matrix(a, a_height * a_width);
		fill_matrix(b, b_height * b_width);
		hipMemcpy(a_device, a, b_height * b_width * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(b_device, b, b_height * b_width * sizeof(float), hipMemcpyHostToDevice);

		clock_t t1 = clock();
		float time;
		matrix_mul_cpu(a, b, c, a_height, a_width, b_width);
		t1 = clock() - t1;
		time_h += t1;

		hipEventRecord(start, 0);
		status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, b_width, a_height, a_width, &alpha, b_device, b_width, a_device,
			a_width, &beta, c_device, b_width);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		time_d += (time / 1000.0);

		sum_h = calculate_sum_cpu(c, a_height * b_width);
		print_matrix(c, a_height, b_width);
		hipMemcpy(c, c_device, a_height * b_width * sizeof(float), hipMemcpyDeviceToHost);
		sum_d = calculate_sum_cpu(c, a_height * b_width);
		print_matrix(c, a_height, b_width);
		if (sum_h == sum_d)
		{
			printf("OK! ");
		}
		printf("cpu sum: %f; cublas sum: %f\n", sum_d);
	}

	float time_h_secs = (float)time_h / CLOCKS_PER_SEC;
	float profit = time_h_secs / time_d;
	printf("profit: %f\n", profit);

	//fclose(stdout);
	status = hipblasDestroy(handle);
	free(a);
	free(b);
	free(c);
	hipFree(a_device);
	hipFree(b_device);
	hipFree(c_device);

    return 0;
}

