#include "hip/hip_runtime.h"

#include <cstdlib>
#include <ctime>
#include <cstdio>
#include <iostream>
#include <hipblas.h>

#define BLOCK_SIZE 32

void matMulCpu(float* a, float* b, float* c, int s1, int s2, int s3)
{
	for (int i = 0; i < s1; i++)
	{
		for (int j = 0; j < s3; j++)
		{
			float ans = 0;
			for (int k = 0; k < s2; k++)
			{
				ans += a[i * s2 + k] * b[k * s3 + j];
			}
			c[i * s3 + j] = ans;
		}
	}
}

__global__ void matMulGpu(float* a, float* b, float* c, int s1, int s2, int s3)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < s1 && j < s3)
	{
		float ans = 0;
		for (int k = 0; k < s2; k++)
		{
			ans += a[i * s2 + k] * b[k * s3 + j];
		}
		c[i * s3 + j] = ans;
	}
}

void fillMat(float* a, int s)
{
	for (int i = 0; i < s; i++)
	{
		a[i] = rand() % 100;
	}
}

float calcSumCpu(float* a, int s)
{
	float ans = 0;
	for (int i = 0; i < s; i++)
	{
		ans += a[i];
	}
	return ans;
}

int main()
{
	//freopen("out.txt", "w", stdout);
	int iterations = 2;
	int size = 1000;
	srand(time(NULL));
	float* a = (float*)malloc(size * size * sizeof(float));
	float* b = (float*)malloc(size * size * sizeof(float));
	float* c = (float*)malloc(size * size * sizeof(float));

	float* a_device;
	float* b_device;
	float* c_device;
	float* c_device_cublas;
	hipMalloc(&a_device, size * size * sizeof(float));
	hipMalloc(&b_device, size * size * sizeof(float));
	hipMalloc(&c_device, size * size * sizeof(float));
	hipMalloc(&c_device_cublas, size * size * sizeof(float));

	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(1 + size / (1 + BLOCK_SIZE), 1 + size / (1 + BLOCK_SIZE));

	float sum_h;
	float sum_d;

	clock_t time_h = 0;
	clock_t time_d = 0;
	float time_d_event = 0;
	float time_d_cublas = 0;

	hipblasHandle_t handle;
	hipblasStatus_t status = hipblasCreate(&handle); 
	float alpha = 1.0f;
	float beta = 0.0f;

	// warmup
	fillMat(a, size * size);
	fillMat(b, size * size);
	hipMemcpy(a_device, a, size * size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b_device, b, size * size * sizeof(float), hipMemcpyHostToDevice);
	matMulGpu<<<grid, block>>>(a_device, b_device, c_device, size, size, size);
	hipDeviceSynchronize();
	status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, size, size, size, &alpha, b, size, a, size,
			&beta, c_device_cublas, size);
	hipDeviceSynchronize();
	// done

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for (int i = 0; i < iterations; i++)
	{
		fillMat(a, size * size);
		fillMat(b, size * size);
		hipMemcpy(a_device, a, size * size * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(b_device, b, size * size * sizeof(float), hipMemcpyHostToDevice);

		clock_t t1 = clock();
		float time;
		matMulCpu(a, b, c, size, size, size);
		t1 = clock() - t1;
		time_h += t1;

		t1 = clock();
		hipEventRecord(start, 0);
		matMulGpu<<<grid, block>>>(a_device, b_device, c_device, size, size, size);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipDeviceSynchronize();
		t1 = clock() - t1;
		time_d += t1;

		hipEventElapsedTime(&time, start, stop);
		time_d_event += (time / 1000.0);

		hipEventRecord(start, 0);
		status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, size, size, size, &alpha, b, size, a, size,
			&beta, c_device_cublas, size);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		time_d_cublas += (time / 1000.0);

		sum_h = calcSumCpu(c, size * size);
		hipMemcpy(c, c_device, size * size * sizeof(float), hipMemcpyDeviceToHost);
		sum_d = calcSumCpu(c, size * size);
		if (sum_h == sum_d)
		{
			printf("OK ");
		}
		printf("host: %f device: %f ", sum_h, sum_d);
		hipMemcpy(c, c_device_cublas, size * size * sizeof(float), hipMemcpyDeviceToHost);
		sum_d = calcSumCpu(c, size * size);
		if (sum_h == sum_d)
		{
			printf("OK ");
		}
		printf("cublas: %f\n", sum_d);
	}

	float time_h_secs = (float)time_h / CLOCKS_PER_SEC;
	float time_d_secs = (float)time_d / CLOCKS_PER_SEC;
	float profit = time_h_secs / time_d_secs;
	float profit_event = time_h_secs / time_d_event;
	float profit_cublas = time_h_secs / time_d_cublas;
	printf("profit: %f profit event: %f profit cublas %f time_h: %f time_d: %f\n\n", profit,
		profit_event, profit_cublas, time_h_secs, time_d_secs);

	//fclose(stdout);
	status = hipblasDestroy(handle);
	free(a);
	free(b);
	free(c);
	hipFree(a_device);
	hipFree(b_device);
	hipFree(c_device);

    return 0;
}
